#include "hip/hip_runtime.h"

#define READ_AND_MAP(i) (make_float2(x[i], x[i]))
#define REDUCE(a, b) (make_float2(min(a.x, b.x), max(a.y, b.y)))



__global__ void max_min_reduction_stage1( global const float *data,
                                      global float2 *out,
                                      int size,
                                      local  float2 *l_data)// local storage 2 float per thread
{
    int group_size =  get_local_size(0);
    int lid = get_local_id(0);
    float2 acc;
    int big_block = group_size * get_num_groups(0);
    int i =  lid + group_size * get_group_id(0);

    if (lid<size)
        acc = read_and_map(lid, data);
    else
        acc = read_and_map(0, data);

    // Linear pre-reduction stage 0

    while (i<size){
      acc = REDUCE(acc, read_and_map(i, data));
      i += big_block;
    }

    // parallel reduction stage 1

    l_data[lid] = acc;
    barrier(CLK_LOCAL_MEM_FENCE);
    for (int block=group_size/2; block>1; block/=2)
        {
            if ((lid < block) && ((lid + block)<group_size)){
                l_data[lid] = REDUCE(l_data[lid], l_data[lid + block]);
            }
            barrier(CLK_LOCAL_MEM_FENCE);
        }
    if (lid == 0)
    {
        if (group_size > 1)
        {
            acc = REDUCE(l_data[0], l_data[1]);
        }
        else
        {
            acc = l_data[0];
        }
        out[get_group_id(0)] = acc;
    }
}


__global__ void max_min_reduction_stage2(
        global const float2 *data2,
        global float2 *maxmin,
        local  float2 *l_data)// local storage 2 float per thread
{
    int lid = get_local_id(0);
    int group_size =  get_local_size(0);
    float2 acc = (float2)(-1.0f, -1.0f);
    if (lid<=group_size)
    {
        l_data[lid] = data2[lid];
    }
    else
    {
        l_data[lid] = acc;
    }

    // parallel reduction stage 2


    barrier(CLK_LOCAL_MEM_FENCE);
    for (int block=group_size/2; block>1; block/=2)
    {
        if ((lid < block) && ((lid + block)<group_size))
        {
            l_data[lid] = REDUCE(l_data[lid], l_data[lid + block]);
        }
        barrier(CLK_LOCAL_MEM_FENCE);

    }

    if (lid == 0 )
    {
        if ( group_size > 1)
        {
            acc = REDUCE(l_data[0], l_data[1]);
        }
        else
        {
            acc = l_data[0];
        }
        maxmin[0] = acc;
    }
}
